#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include "ppm.cpp"

// Fonction pour calculer l'angle entre deux points
__device__ float calculateAngle(float xc, float yc, float xa, float ya, float xb, float yb)
{
    return atan2f(yb - yc, xb - xc);
}

// Kernel CUDA pour le calcul de la visibilité
__global__ void naive_viewtest(const uint8_t *heightmap, int width, int height, int centerX, int centerY, uint8_t *result)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float xc = x - centerX;
        float yc = y - centerY;

        for (int i = 0; i < width; i++)
        {
            for (int j = 0; j < height; j++)
            {
                float angle = calculateAngle(xc, yc, i - centerX, j - centerY);

                // Vérifiez la condition de visibilité
                if (angle < calculateAngle(xc, yc, x - centerX, y - centerY))
                {
                    // Le pixel est visible, marquez-le d'une certaine manière dans le résultat
                    result[x + y * width] = 255; // Marquez en blanc (ajustez selon vos besoins)
                }
            }
        }
    }
}

int main()
{
    // Chargez votre carte de hauteur (replacez le chemin du fichier PPM)
    los::Heightmap heightmap("Test/test_leger.ppm");

    // Obtenez les données de la carte de hauteur
    uint8_t *heightmapData = heightmap.getPtr();

    // Dimensions de la grille et du bloc
    dim3 blockSize(16, 16);
    dim3 gridSize((heightmap.getWidth() + blockSize.x - 1) / blockSize.x, (heightmap.getHeight() + blockSize.y - 1) / blockSize.y);

    // Résultat du calcul de visibilité
    uint8_t *visibilityResult = new uint8_t[heightmap.getSize()];

    // Appel du kernel CUDA
    naive_viewtest<<<gridSize, blockSize>>>(heightmapData, heightmap.getWidth(), heightmap.getHeight(), centerX, centerY, visibilityResult);

    // Attendez que tous les threads se terminent
    hipDeviceSynchronize();

    // Sauvegardez le résultat dans un nouveau fichier PPM (replacez le chemin de sortie)
    los::Heightmap visibilityMap(heightmap.getWidth(), heightmap.getHeight());
    for (std::size_t i = 0; i < heightmap.getSize(); i++)
    {
        visibilityMap.setPixel(i % heightmap.getWidth(), i / heightmap.getWidth(), visibilityResult[i]);
    }
    visibilityMap.saveTo("/");

    // Libérez la mémoire
    delete[] visibilityResult;

    return 0;
}
